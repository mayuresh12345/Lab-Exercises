#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#define N 100

__global__ void CUDAStrCopy(char *str, char *sub, int *len2)
{
	int start = blockIdx.x;
	int end = start + *len2;
	for(int i = start; i < end; i++)
	{
		if(str[i] != sub[i - start])
			break;
		else if(i == end - 1)
			printf("Found at %d\n", (i - start));
	}
}

int main()
{
	char str[N];
	char sub[N];
	char *pstr, *psub;
	int *plen2;

	printf("Enter a string: ");
	scanf("%s", str);
	printf("Enter the substring: ");
	scanf("%s", sub);

	int len1 = strlen(str);
	int len2 = strlen(sub);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMalloc((void**)&pstr, N * sizeof(char));
	hipMalloc((void**)&psub, len2 * sizeof(char));
	hipMalloc((void**)&plen2, sizeof(int));
	hipMemcpy(pstr, str, N * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(psub, sub, len2 * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(plen2, &len2, sizeof(int), hipMemcpyHostToDevice);

	CUDAStrCopy<<<N - len2, 1>>>(pstr, psub, plen2);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	int i;
	/*
		printf("Value of C in host after kernel execution\n");
		for(int i = 0; i < N; i++)
			printf("%c\n", C[i]);
	*/
	printf("Time taken = %f", elapsedTime);
	hipFree(pstr);
	hipFree(psub);
	hipFree(plen2);
	printf("\n");

	return 0;
}
