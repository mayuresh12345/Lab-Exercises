#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define N 1024

__global__ void CUDABinary(char *A, int *L)
{
	int i = threadIdx.x;

	int dec = (int)A[i];
	int j = 1, rem[10] = {0};

	while(dec != 0)
	{
		rem[j++] = dec % 2;
		dec /= 2;
	}

	int bin = 0;
	while(j != 0)
	{
		bin = bin * 10 + rem[j--];
	}

	printf("%d  ", bin);
	//printf("%s\n", C[i]);
}

int main(int argc, char const *argv[])
{
	char A[N];
	//char C[N];

	char *pA;
	int *pL;

	printf("Enter string(A): ");
	scanf("%[^\n]%*c", A);

	//printf("C = \n");>>>(p

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&pA, N * sizeof(char));
	hipMalloc((void**)&pL, sizeof(int));
	//cudaMalloc((void**)&pC, N * sizeof(char));

	int L = strlen(A);

	hipMemcpy(pA, A, N * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pL, &L, sizeof(int), hipMemcpyHostToDevice);

	hipError_t error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
	}

	printf("Binary values:\n");

	CUDABinary<<<1, L>>>(pA, pL);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("CUDA Error2: %s\n", hipGetErrorString(error));
	}

	hipMemcpy(A, pA, N * sizeof(char), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	//printf("Value of C in Host:\n");

	//printf("Reversed A: %s\n", A);
	/*for (int i = 0; i < N; ++i)
	{
		printf("%c\n", C[i]);
	}*/

	printf("\nTime Taken = %f\n", elapsedTime);

	hipFree(pA);
	hipFree(pL);
	//cudaFree(pC);

	return 0;
}
