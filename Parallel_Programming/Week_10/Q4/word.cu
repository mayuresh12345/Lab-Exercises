#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDASubstring(char *A, char *C, int *sL, int *pFound)
{
	int id = threadIdx.x;
	if(id == 0 || A[id-1] == ' ')
	{
		int fMatch = 1;
		for (int i = 0; i < *sL; i++)
		{
			if (A[id + i] != C[i])
				fMatch = 0;
		}
		if (fMatch == 1)
		{
			atomicAdd(pFound, 1);
		}
	}
}

int main(int argc, char const *argv[])
{
	char A[N];
	char C[N];

	char *pA, *pC;
	//int *pL;

	printf("Enter string(A): ");
	scanf("%[^\n]%*c", A);

	printf("Enter substring(C): ");
	scanf("%[^\n]%*c", C);

	int L = strlen(A);
	int sL = strlen(C);
	int *pFound, *psL;
	//printf("C = \n");

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMalloc((void**)&pA, L * sizeof(char));
	hipMalloc((void**)&pC, sL * sizeof(char));
	hipMalloc((void**)&psL, sizeof(int));
	hipMalloc((void**)&pFound, sizeof(int));

	int t = 0;
	//cudaMemcpy(pA, A, L * sizeof(char), cudaMemcpyHostToDevice);
	hipMemcpy(pA, A, L * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, sL * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(psL, &sL, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pFound, &t, sizeof(int), hipMemcpyHostToDevice);

	hipError_t error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("CUDA Error1: %s\n", hipGetErrorString(error));
	}

	CUDASubstring<<<1, L>>>(pA, pC, psL, pFound);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("CUDA Error2: %s\n", hipGetErrorString(error));
	}

	int found;

	hipMemcpy(&found, pFound, sizeof(int), hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	/*printf("Value of C in Host:\n");

	for (int i = 0; i < N; ++i)
	{
		printf("%c\n", C[i]);
	}*/

	if(found != 0)
		printf("Found %d times\n", found);
	else
		printf("Not Found\n");

	printf("Time Taken = %f\n", elapsedTime);

	hipFree(pA);
	hipFree(pC);

	return 0;
}
