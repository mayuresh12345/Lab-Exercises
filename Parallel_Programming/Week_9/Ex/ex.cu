
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}

int main()
{
	int a, b, c;

	// host copies of variables a, b & c
	int *d_a, *d_b, *d_c;

	// device copies of variables a, b & c
	int size = sizeof(int);

	// Allocate space for device copies a, b & c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	// Setup input values
	a = 4;
	b = 8;

	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<1, 1>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// print result
	printf("%d + %d = %d\n", a, b, c);

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
