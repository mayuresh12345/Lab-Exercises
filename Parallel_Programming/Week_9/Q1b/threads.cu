
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 5
__global__ void add(int *a, int *b, int *c)
{
	int tid = threadIdx.x;

//	if(tid < N)
		c[tid] = a[tid] + b[tid];
}

int main()
{
//	int *a, *b, *c;
	int a[N], b[N], c[N];

	// host copies of variables a, b & c
	int *d_a, *d_b, *d_c;

	// device copies of variables a, b & c
	int size = sizeof(int) * N;

//	a = (int *)malloc(sizeof(int) * N);
//	b = (int *)malloc(sizeof(int) * N);
//	c = (int *)malloc(sizeof(int) * N);

	// Allocate space for device copies a, b & c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	// Setup input values
	printf("Enter values for a: ");
	for(int i = 0; i < N; i++)
		scanf("%d", &a[i]);
	printf("Enter values for b: ");
		for(int i = 0; i < N; i++)
			scanf("%d", &b[i]);

	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<1, N>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// print result
	for(int i = 0; i < N; i++)
		printf("%d + %d = %d\n", a[i], b[i], c[i]);

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
