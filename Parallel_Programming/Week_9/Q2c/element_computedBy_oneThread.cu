
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 3
#define M 2

__global__ void add(int *A, int *B, int *C)
{
//    int bid = blockIdx.x;
    int tid = threadIdx.x;
    
    printf("(%d)", tid);
    C[tid] = A[tid] + B[tid];    
}

int main()
{
//	int *a, *b, *c;
	int a[M * N], b[M * N], c[M * N];

	// host copies of variables a, b & c
	int *d_a, *d_b, *d_c;

	// device copies of variables a, b & c
	int size = sizeof(int) * M * N;

//	a = (int *)malloc(sizeof(int) * N);
//	b = (int *)malloc(sizeof(int) * N);
//	c = (int *)malloc(sizeof(int) * N);

	// Allocate space for device copies a, b & c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
  
  int t = 1;
	// Setup input values
	printf("Enter values for a: ");
	for(int i = 0; i < M; i++)
		for(int j = 0; j < N; j++)
      a[i * N + j] = t++;
  
  t = 1;
	printf("Enter values for b: ");
	for(int i = 0; i < M; i++)
		for(int j = 0; j < N; j++)
      b[i * N + j] = t++;

	// Copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<1, M*N>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	// print result
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
			{
          printf("%d + %d = %d\n", a[i * N + j], b[i * N + j], c[i * N + j]);
      }
      
	}

	// Cleanup
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}